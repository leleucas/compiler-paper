#include "hip/hip_runtime.h"

#include "elena_int.h"
extern "C" __global__ void coderop_0(float* __restrict__ Var0, float* __restrict__ Var1, float* __restrict__ Var9) {
    const uint64_t iter2_iter3_fused = ((blockIdx.x * 64) + threadIdx.x);
    const uint64_t iter3 = (iter2_iter3_fused % 5);
    const uint64_t iter2 = (iter2_iter3_fused / 5);
    if ((iter2 < 256)) {
        if ((iter3 < 5)) {
            Var9[(iter3 + (5 * iter2))] = ((Var0[((iter3 * 2) + (9 * iter2))] + Var1[((iter3 * 2) + (9 * iter2))]) - Var0[((iter3 * 2) + (9 * iter2))]);
        }
    }
}

