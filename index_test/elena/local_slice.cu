#include "hip/hip_runtime.h"

#include "elena_int.h"
extern "C" __global__ void coderop_0(float* __restrict__ Var0, float* __restrict__ Var1, float* __restrict__ Var7) {
    const uint64_t iter2_iter3_fused = ((blockIdx.x * 64) + threadIdx.x);
    const uint64_t iter3 = (iter2_iter3_fused % 4);
    const uint64_t iter2 = (iter2_iter3_fused / 4);
    if ((iter2 < 256)) {
        if ((iter3 < 4)) {
            Var7[(iter3 + (4 * iter2))] = ((Var0[((iter3 * 2) + (9 * iter2))] + Var1[((iter3 * 2) + (9 * iter2))]) - (Var0[(((iter3 * 2) + (9 * iter2)) + 1)] + Var1[(((iter3 * 2) + (9 * iter2)) + 1)]));
        }
    }
}

