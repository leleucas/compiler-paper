#include "hip/hip_runtime.h"

#include "elena_int.h"
extern "C" __global__ void coderop_0(float* __restrict__ Var1, float* __restrict__ Var2, float* __restrict__ Var0, float* __restrict__ Var6) {
    const uint64_t iter1_iter2_fused = ((blockIdx.x * 64) + threadIdx.x);
    const uint64_t iter2 = (iter1_iter2_fused % 4);
    const uint64_t iter1 = (iter1_iter2_fused / 4);
    if ((iter1 < 256)) {
        if ((iter2 < 4)) {
            Var6[(iter2 + (4 * iter1))] = ((Var1[((iter2 * 0) + iter1)] + Var2[((iter2 * 0) + iter1)]) - Var0[(iter2 + (4 * iter1))]);
        }
    }
}

