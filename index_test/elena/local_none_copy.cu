#include "hip/hip_runtime.h"

#include "elena_int.h"
extern "C" __global__ void coderop_0(float* __restrict__ Var1, float* __restrict__ Var2, float* __restrict__ Var0, float* __restrict__ Var9) {
    const uint64_t iter4_iter5_fused = ((blockIdx.x * 64) + threadIdx.x);
    const uint64_t iter5 = (iter4_iter5_fused % 1);
    const uint64_t iter4 = iter4_iter5_fused;
    if ((iter4 < 256)) {
        if ((iter5 < 1)) {
            Var9[(iter5 + iter4)] = (((1 == 1) ? Var0[iter4] : (Var1[((iter5 * 0) + iter4)] + Var2[((iter5 * 0) + iter4)])) * Var0[(iter5 + iter4)]);
        }
    }
}

