#include "hip/hip_runtime.h"

#include "elena_int.h"
extern "C" __global__ void coderop_1(float* __restrict__ Var1, float* __restrict__ Var0, float* __restrict__ Var2, float* __restrict__ Var8) {
    const uint64_t iter10_iter11_fused = ((blockIdx.x * 64) + threadIdx.x);
    const uint64_t iter11 = (iter10_iter11_fused % 1);
    const uint64_t iter10 = iter10_iter11_fused;
    if ((iter10 < 3000)) {
        if ((iter11 < 1)) {
            Var2[(iter11 + iter10)] = Var1[(iter11 + iter10)];
        }
    }
    const uint64_t iter13 = ((blockIdx.x * 64) + threadIdx.x);
    if ((iter13 < 3000)) {
        Var8[iter13] = ((Var0[((iter13 * 4) + 1)] + Var0[((iter13 * 4) + 3)]) * 0.5);
    }
}

