#include "hip/hip_runtime.h"

#include "elena_int.h"
extern "C" __global__ void coderop_0(float* __restrict__ Var1, float* __restrict__ Var3, float* __restrict__ Var2, float* __restrict__ Var0, float* __restrict__ Var5, float* __restrict__ Var14, float* __restrict__ Var15, float* __restrict__ Var21) {
    const uint64_t iter2_iter3_fused = ((blockIdx.x * 64) + threadIdx.x);
    const uint64_t iter3 = (iter2_iter3_fused % 4);
    const uint64_t iter2 = (iter2_iter3_fused / 4);
    if ((iter2 < 3000)) {
        if ((iter3 < 4)) {
            Var5[(iter3 + (4 * iter2))] = ((Var1[(iter3 + (4 * iter2))] * Var3[(iter3 + (0 * iter2))]) + Var2[(iter3 + (0 * iter2))]);
        }
    }
    const uint64_t iter4_iter5_fused = ((blockIdx.x * 64) + threadIdx.x);
    const uint64_t iter5 = (iter4_iter5_fused % 1);
    const uint64_t iter4 = iter4_iter5_fused;
    if ((iter4 < 3000)) {
        if ((iter5 < 1)) {
            Var14[(iter5 + iter4)] = fmin((float)fmax((float)((Var1[(((iter5 * 2) + (4 * iter4)) + 2)] * Var3[(((iter5 * 2) + (0 * iter4)) + 2)]) + Var2[(((iter5 * 2) + (0 * iter4)) + 2)]), (float)-4.13517), (float)4.13517);
        }
    }
    const uint64_t iter6_iter7_fused = ((blockIdx.x * 64) + threadIdx.x);
    const uint64_t iter7 = (iter6_iter7_fused % 1);
    const uint64_t iter6 = iter6_iter7_fused;
    if ((iter6 < 3000)) {
        if ((iter7 < 1)) {
            Var15[(iter7 + iter6)] = fmin((float)fmax((float)((Var1[((iter7 + (4 * iter6)) + 3)] * Var3[((iter7 + (0 * iter6)) + 3)]) + Var2[((iter7 + (0 * iter6)) + 3)]), (float)-4.13517), (float)4.13517);
        }
    }
    const uint64_t iter9 = ((blockIdx.x * 64) + threadIdx.x);
    if ((iter9 < 3000)) {
        Var21[iter9] = ((Var0[(iter9 * 4)] + Var0[((iter9 * 4) + 2)]) * 0.5);
    }
}

