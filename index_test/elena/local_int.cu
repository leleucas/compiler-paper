#include "hip/hip_runtime.h"

#include "elena_int.h"
extern "C" __global__ void coderop_0(float* __restrict__ Var0, float* __restrict__ Var1, float* __restrict__ Var2, float* __restrict__ Var6) {
    const uint64_t iter2 = ((blockIdx.x * 64) + threadIdx.x);
    if ((iter2 < 256)) {
        Var6[iter2] = ((Var0[((iter2 * 4) + 2)] + Var1[((iter2 * 4) + 2)]) - Var2[iter2]);
    }
}

