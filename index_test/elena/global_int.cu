#include "hip/hip_runtime.h"

#include "elena_int.h"
extern "C" __global__ void coderop_0(float* __restrict__ Var0, float* __restrict__ Var1, float* __restrict__ Var5) {
    const uint64_t iter1 = ((blockIdx.x * 64) + threadIdx.x);
    if ((iter1 < 256)) {
        Var5[iter1] = ((Var0[((iter1 * 4) + 1)] + Var1[iter1]) - Var1[iter1]);
    }
}

