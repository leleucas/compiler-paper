#include "hip/hip_runtime.h"

#include "elena_int.h"
extern "C" __global__ void coderop_0(float* __restrict__ Var0, float* __restrict__ Var1, float* __restrict__ Var2, float* __restrict__ Var7) {
    const uint64_t iter5_iter6_fused = ((blockIdx.x * 64) + threadIdx.x);
    const uint64_t iter6 = (iter5_iter6_fused % 4);
    const uint64_t iter5 = (iter5_iter6_fused / 4);
    if ((iter5 < 256)) {
        if ((iter6 < 4)) {
            Var7[(iter6 + (4 * iter5))] = (((Var0[(iter6 + (4 * iter5))] + Var1[(iter6 + (4 * iter5))]) - Var1[(iter6 + (4 * iter5))]) + (((1 == 1) && (iter6 == 3)) ? Var2[iter5] : (Var0[(iter6 + (4 * iter5))] + Var1[(iter6 + (4 * iter5))])));
        }
    }
}

